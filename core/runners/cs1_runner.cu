#include "hip/hip_runtime.h"
#ifndef cs1_runner_cu
#define cs1_runner_cu


#include "cs1_runner.cuh"
#include <fstream>


report_map Runners::CS1Runner::naive(const double confidence, const bool save_images, const std::string &data_path)
{
    report_map report;

    report.insert({"mask_length", parameters->mask_length});
    report.insert({"value_length", parameters->value_length});
    report.insert({"address_length", parameters->address_length});
    report.insert({"cells_count", parameters->cells_count});
    report.insert({"image_count", parameters->image_count});
    report.insert({"images_read", parameters->images_read});
    report.insert({"block_count", parameters->block_count});
    report.insert({"threads_per_block", parameters->threads_per_block});
    report.insert({"labels_count", parameters->labels_count});
    report.insert({"target_count", parameters->target_count});
    report.insert({"bits_per_num", parameters->bits_per_num});

    int rows = parameters->target_count;
    int columns = 4*parameters->labels_count;
    bool* transformation = (bool*) malloc(rows*columns*sizeof(bool));
    bool* cuda_transformation;
    hipMalloc((void**)&cuda_transformation, rows*columns*sizeof(bool));
    check_errors<int>("hipMalloc/cuda_transformation");

    generate_small_random_matrix<<<parameters->block_count, parameters->threads_per_block>>>
                                (rows, columns, cuda_transformation);
    hipMemcpy(transformation, cuda_transformation, rows*columns*sizeof(bool), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    check_errors<int>("hipMemcpy/transformation");
    uint m = rows*parameters->image_count;

    typedef int SUM_TYPE;

    ulong result_bytes = m*sizeof(SUM_TYPE);

    SUM_TYPE* cuda_transformed;
    hipMalloc((void**)&cuda_transformed, result_bytes);
    check_errors<int>("hipMalloc/cuda_transformed");
    hipMemset(cuda_transformed, (SUM_TYPE)0, result_bytes);
    hipDeviceSynchronize();

    SUM_TYPE* transformed = (SUM_TYPE*) malloc(result_bytes);
    check_errors<int>("hipMemset/cuda_transformed");

    bool* cuda_data;
    hipMalloc((void**)&cuda_data, columns*parameters->image_count*sizeof(bool));
    check_errors<int>("hipMalloc/cuda_data");
    hipMemcpy(cuda_data, data, columns*parameters->image_count*sizeof(bool), hipMemcpyHostToDevice);
    check_errors<int>("hipMemcpy/cuda_data");

    uint tc = parameters->block_count*parameters->threads_per_block;
    hipDeviceSynchronize();
    mult_matrix<SUM_TYPE><<<parameters->block_count, parameters->threads_per_block>>>
                (cuda_transformation, cuda_data, cuda_transformed, rows, columns, parameters->image_count, tc);
    hipDeviceSynchronize();
    check_errors<int>("mult_matrix");
    hipMemcpy(transformed, cuda_transformed, result_bytes, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    check_errors<int>("cudaMemcpyResult/transformed2");
    std::cout << std::endl;
    auto max = transformed[0];
    auto max_i = 0;
    auto min = transformed[0];
    auto min_i = 0;
    for (int i = 1; i < m; i++)
    {
        auto el = transformed[i];
        if (min > el)
        {
            min_i = i;
            min = el;
        }
        if (max < el)
        {
            max_i = i;
            max = el;
        }
    }
    std::cout <<std::endl;
    std::cout << "max=" << max << " max_i=" << max_i << std::endl;
    std::cout << "min=" << min << " min_i=" << min_i << std::endl << std::endl;

    std::vector<short*> images(parameters->image_count);
    long pos = 0;
    long neg = 0;
    long z = 0;
    for(int i = 0; i < parameters->image_count; i++)
    {
        auto image = (short*) malloc((rows * sizeof(short)));
        for(int j = 0; j < rows; j++)
        {
            auto el = transformed[j*parameters->image_count+i];
            image[j] = el;
            if (el > 0)
                pos++;
            if (el < 0)
                neg++;
            if (el == 0)
                z++;
        }
        images[i] = image;
    }

    hipFree(cuda_transformation);
    hipFree(cuda_transformed);

    SDM_CS1<int, short, short, short> sdm(parameters->mask_length, parameters->address_length,
                                            parameters->value_length, parameters->cells_count, parameters->block_count,
                                            parameters->threads_per_block);

    long write_time_start = clock();
    std::cout << "Started writing ";
    int act_zero = 0;
    std::vector<int> acts(parameters->image_count);
    for(int i = 0; i < parameters->images_read; i++)
    {
        short* image = images[i];
        //auto* image_noisy = noise(image, 150, 10, i);
        int act = sdm.write(image);
        act_zero += (act == 0);
        acts[i] = act;
        if ((i+1) % 1000 == 0)
            std::cout << (i+1) << " ";
        //free(image_noisy);
//        for (int j = 0; j < 150; j++)
//            std::cout << image[j] << ",";
    }
    std::cout << std::endl ;
    //sdm.print_state();
    long write_time = clock() - write_time_start;

    double sum_l1 = 0;
    double sum_l1_2 = 0;
    long read_time_start = clock();
    //sdm.print_state();
    std::cout << "Started reading" << " ";
    double sum_l1_arr = 0;

    double max_l1 = 0;
    double max_l1_ind = -1;

    double min_l1 = 1e12;
    double min_l1_ind = -1;

    double avg_l1_r = 0;
    double avg_l1_f = 0;
    double avg_l1_c = 0;
    int read_zeros = 0;
    std::ofstream restored;
    restored.open("C:\\Development\\PhD\\Analysis\\data\\restored_K_" + std::to_string(parameters->mask_length) +
                    "_I_" + std::to_string(parameters->images_read) + ".txt");
    for(int i = 0; i < parameters->images_read; i++)
    {
        auto el = images[i];
        double l1_arr = 0;
        double* remembered = sdm.read(el);
        double l1 = 0;
        double l1_r = 0;
        double l1_f = 0;
        double l1_c = 0;
        std::vector<double> rem_arr(parameters->value_length);
        std::vector<short> img_arr(parameters->value_length);
        bool is_zeros = true;
        for (int j = 0; j < parameters->value_length; j++)
        {
            double rem = remembered[j];
            restored << rem << ",";
            if (abs(rem) > 1e-6)
                is_zeros = false;
            //double rem2 = remembered2[j];
            rem_arr[j] = rem;
            //rem2_arr[j] = rem2;
            auto elj = el[j];
            img_arr[j] = elj;
            double elj_r = round(rem);
            double elj_f = floor(rem);
            double elj_c = ceil(rem);
            l1_r += abs(elj_r - elj);
            l1_f += abs(elj_f - elj);
            l1_c += abs(elj_c - elj);
            l1 += abs(rem - elj);
            //l1_2 += abs(rem2 - elj);
        }
        restored << std::endl;
        if (is_zeros)
        {
            read_zeros += 1;
            continue;
        }
        if (max_l1 < l1)
        {
            max_l1 = l1;
            max_l1_ind = i;
        }
        if (min_l1 > l1)
        {
            min_l1 = l1;
            min_l1_ind = i;
        }

        sum_l1 += l1;
        l1_arr += l1;
        sum_l1_arr += l1_arr;
        avg_l1_r += l1_r / parameters->images_read;
        avg_l1_f += l1_f / parameters->images_read;
        avg_l1_c += l1_c / parameters->images_read;
        //std::cout << dist << "|";
//        for(int j = 0; j < parameters->value_length; j++)
//        {
//            std::cout << remembered[j] << "|";
//        }
//        std::cout << std::endl;
//        std::cout << dist << "|";
        free(remembered);
        if ((i+1) % 1000 == 0)
            std::cout << (i+1) << " ";
    }
    std::cout << std::endl;
    long read_time = clock() - read_time_start;
    //restored.close();

    report.insert({"act_zero", act_zero});
    report.insert({"avg_l1", sum_l1 / parameters->images_read});
    report.insert({"avg_l1_r", avg_l1_r});
    report.insert({"avg_l1_f", avg_l1_f});
    report.insert({"avg_l1_c", avg_l1_c});
    report.insert({"max_l1", max_l1});
    report.insert({"max_l1_ind", max_l1_ind});
    report.insert({"min_l1", min_l1});
    report.insert({"min_l1_ind", min_l1_ind});
    report.insert({"mae", sum_l1_arr / parameters->images_read / rows});
    report.insert({"avg_read_time", (double)read_time / parameters->images_read});
    report.insert({"avg_write_time", (double)write_time / parameters->images_read});
    report.insert({"read_zeros", read_zeros});
//    report.insert({"min_activations", sdm.get_min_activations()});
//    report.insert({"max_activations", sdm.get_max_activations()});
//    report.insert({"activated_cells_count", sdm.get_activations_num()});

    //sdm.~SDM_CS1();

    return report;
}

#endif //cs1_runner_cu
