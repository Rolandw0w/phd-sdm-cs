#include "hip/hip_runtime.h"
#ifndef cs1_runner_cu
#define cs1_runner_cu


#include "cs1_runner.cuh"
#include <fstream>


report_map Runners::CS1Runner::naive(const std::string& data_path, const std::string& output_path)
{
    report_map report;

    report.insert({"mask_length", parameters->mask_length});
    report.insert({"value_length", parameters->value_length});
    report.insert({"address_length", parameters->address_length});
    report.insert({"cells_count", parameters->cells_count});
    report.insert({"image_count", parameters->image_count});
    report.insert({"images_read", parameters->images_read});
    report.insert({"block_count", parameters->block_count});
    report.insert({"threads_per_block", parameters->threads_per_block});
    report.insert({"labels_count", parameters->labels_count});
    report.insert({"target_count", parameters->target_count});
    report.insert({"bits_per_num", parameters->bits_per_num});

    int rows = parameters->target_count;
    int columns = 4*parameters->labels_count;
    bool* transformation = (bool*) malloc(rows*columns*sizeof(bool));
    bool* cuda_transformation;

    cuda_malloc(&cuda_transformation, rows*columns);

    kernel_decorator(
            generate_small_random_matrix<bool>,
            parameters->block_count, parameters->threads_per_block, true,
            rows, columns, cuda_transformation
    );

    cuda_memcpy_from_gpu(transformation, cuda_transformation, rows*columns);

    uint transformation_size = rows*parameters->image_count;

    std::ofstream transformation_file;
    auto transformation_file_path = output_path + "/cs1_matrix_K_" + std::to_string(parameters->mask_length) +
                                    "_I_" + std::to_string(parameters->images_read) + ".csv";
    transformation_file.open(transformation_file_path);

    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < columns; j++)
        {
            auto ind = i*columns + j;
            auto el = transformation[ind];
            int to_write = 2*el - 1;
            auto sep = (j == columns - 1) ? "\n" : ",";
            transformation_file << to_write << sep;
        }
    }
    transformation_file.close();

    typedef int SUM_TYPE;

    SUM_TYPE* cuda_transformed;

    cuda_malloc(&cuda_transformed, transformation_size);
    cuda_memset(cuda_transformed, (SUM_TYPE)0, transformation_size);

    auto transformed = (SUM_TYPE*) malloc(transformation_size*sizeof(SUM_TYPE));

    bool* cuda_data;
    cuda_malloc(&cuda_data, columns*parameters->image_count);
    cuda_memcpy_to_gpu(cuda_data, data, columns*parameters->image_count);

    uint thread_count = parameters->block_count*parameters->threads_per_block;
    kernel_decorator(
            mult_matrix<SUM_TYPE>,
            parameters->block_count, parameters->threads_per_block, true,
            cuda_transformation, cuda_data, cuda_transformed, rows, columns, parameters->image_count, thread_count
    );

    cuda_memcpy_from_gpu(transformed, cuda_transformed, transformation_size);

    auto max = transformed[0];
    auto max_i = 0;
    auto min = transformed[0];
    auto min_i = 0;
    for (int i = 1; i < transformation_size; i++)
    {
        auto el = transformed[i];
        if (min > el)
        {
            min_i = i;
            min = el;
        }
        if (max < el)
        {
            max_i = i;
            max = el;
        }
    }
    std::cout <<std::endl;
    std::cout << "max=" << max << " max_i=" << max_i << std::endl;
    std::cout << "min=" << min << " min_i=" << min_i << std::endl << std::endl;

    std::vector<short*> images(parameters->image_count);
    long pos = 0;
    long neg = 0;
    long z = 0;
    for(int i = 0; i < parameters->image_count; i++)
    {
        auto image = (short*) malloc((rows * sizeof(short)));
        for(int j = 0; j < rows; j++)
        {
            auto el = transformed[j*parameters->image_count+i];
            image[j] = el;
            if (el > 0)
                pos++;
            if (el < 0)
                neg++;
            if (el == 0)
                z++;
        }
        images[i] = image;
    }

    cuda_free(cuda_transformation);
    cuda_free(cuda_transformed);

    SDM_CS1<int, short, short, short> sdm(parameters->mask_length, parameters->address_length,
                                            parameters->value_length, parameters->cells_count, parameters->block_count,
                                            parameters->threads_per_block);

    long write_time_start = clock();
    std::cout << "Started writing ";
    int act_zero = 0;
    std::vector<int> acts(parameters->image_count);
    for(int i = 0; i < parameters->images_read; i++)
    {
        short* image = images[i];
        int act = sdm.write(image);
        act_zero += (act == 0);
        acts[i] = act;
        if ((i+1) % 1000 == 0)
            std::cout << (i+1) << " ";
    }
    //sdm.print_state();
    std::cout << std::endl ;
    long write_time = clock() - write_time_start;

    double sum_l1 = 0;
    long read_time_start = clock();
    std::cout << "Started reading ";
    double sum_l1_arr = 0;

    double max_l1 = 0;
    double max_l1_ind = -1;

    double min_l1 = 1e12;
    double min_l1_ind = -1;

    double avg_l1_r = 0;
    double avg_l1_f = 0;
    double avg_l1_c = 0;
    int read_zeros = 0;
    std::ofstream restored;
    restored.open(output_path + "/cs1_noisy_K_" + std::to_string(parameters->mask_length) +
                    "_I_" + std::to_string(parameters->images_read) + ".csv");
    for(int i = 0; i < parameters->images_read; i++)
    {
        auto el = images[i];
        double l1_arr = 0;
        double* remembered = sdm.read(el);
        double l1 = 0;
        double l1_r = 0;
        double l1_f = 0;
        double l1_c = 0;
        std::vector<double> rem_arr(parameters->value_length);
        std::vector<short> img_arr(parameters->value_length);
        bool is_zeros = true;
        for (int j = 0; j < parameters->value_length; j++)
        {
            double rem = remembered[j];
            //std::cout << rem << " ";
            auto sep = (j == parameters->value_length - 1) ? "\n" : ",";
            restored << rem << sep;
            if (abs(rem) > 1e-6)
                is_zeros = false;
            rem_arr[j] = rem;
            auto elj = el[j];
            img_arr[j] = elj;
            double elj_r = round(rem);
            double elj_f = floor(rem);
            double elj_c = ceil(rem);
            l1_r += abs(elj_r - elj);
            l1_f += abs(elj_f - elj);
            l1_c += abs(elj_c - elj);
            l1 += abs(rem - elj);
        }
        if (is_zeros)
        {
            read_zeros += 1;
            continue;
        }
        if (max_l1 < l1)
        {
            max_l1 = l1;
            max_l1_ind = i;
        }
        if (min_l1 > l1)
        {
            min_l1 = l1;
            min_l1_ind = i;
        }

        sum_l1 += l1;
        l1_arr += l1;
        sum_l1_arr += l1_arr;
        avg_l1_r += l1_r / parameters->images_read;
        avg_l1_f += l1_f / parameters->images_read;
        avg_l1_c += l1_c / parameters->images_read;
        free(remembered);
        if ((i+1) % 1000 == 0)
            std::cout << (i+1) << " ";
    }
    std::cout << std::endl;
    long read_time = clock() - read_time_start;
    restored.close();

    report.insert({"act_zero", act_zero});
    report.insert({"avg_l1", sum_l1 / parameters->images_read});
    report.insert({"avg_l1_r", avg_l1_r});
    report.insert({"avg_l1_f", avg_l1_f});
    report.insert({"avg_l1_c", avg_l1_c});
    report.insert({"max_l1", max_l1});
    report.insert({"max_l1_ind", max_l1_ind});
    report.insert({"min_l1", min_l1});
    report.insert({"min_l1_ind", min_l1_ind});
    report.insert({"mae", sum_l1_arr / parameters->images_read / rows});
    report.insert({"avg_read_time", (double)read_time / parameters->images_read});
    report.insert({"avg_write_time", (double)write_time / parameters->images_read});
    report.insert({"read_zeros", read_zeros});
//    report.insert({"min_activations", sdm.get_min_activations()});
//    report.insert({"max_activations", sdm.get_max_activations()});
//    report.insert({"activated_cells_count", sdm.get_activations_num()});

    return report;
}


report_map Runners::CS1Runner::noisy(const std::string& data_path, const std::string& output_path)
{
    report_map report;

    report.insert({"mask_length", parameters->mask_length});
    report.insert({"value_length", parameters->value_length});
    report.insert({"address_length", parameters->address_length});
    report.insert({"cells_count", parameters->cells_count});
    report.insert({"image_count", parameters->image_count});
    report.insert({"images_read", parameters->images_read});
    report.insert({"block_count", parameters->block_count});
    report.insert({"threads_per_block", parameters->threads_per_block});
    report.insert({"labels_count", parameters->labels_count});
    report.insert({"target_count", parameters->target_count});
    report.insert({"bits_per_num", parameters->bits_per_num});

    int rows = parameters->target_count;
    int columns = 4*parameters->labels_count;
    bool* transformation = (bool*) malloc(rows*columns*sizeof(bool));
    bool* cuda_transformation;

    cuda_malloc(&cuda_transformation, rows*columns);

    kernel_decorator(
            generate_small_random_matrix<bool>,
            parameters->block_count, parameters->threads_per_block, true,
            rows, columns, cuda_transformation
    );

    cuda_memcpy_from_gpu(transformation, cuda_transformation, rows*columns);

    uint transformation_size = rows*parameters->image_count;

    std::ofstream transformation_file;
    auto transformation_file_path = output_path + "/cs1_noisy_matrix_K_" + std::to_string(parameters->mask_length) +
                                    "_I_" + std::to_string(parameters->images_read) + ".csv";
    transformation_file.open(transformation_file_path);

    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < columns; j++)
        {
            auto ind = i*columns + j;
            auto el = transformation[ind];
            int to_write = 2*el - 1;
            auto sep = (j == columns - 1) ? "\n" : ",";
            transformation_file << to_write << sep;
        }
    }
    transformation_file.close();

    typedef int SUM_TYPE;

    SUM_TYPE* cuda_transformed;
    cuda_malloc(&cuda_transformed, transformation_size);
    cuda_memset(cuda_transformed, (SUM_TYPE)0, transformation_size);

    SUM_TYPE* cuda_transformed_noisy;
    cuda_malloc(&cuda_transformed_noisy, transformation_size);
    cuda_memset(cuda_transformed_noisy, (SUM_TYPE)0, transformation_size);

    auto transformed = (SUM_TYPE*) malloc(transformation_size*sizeof(SUM_TYPE));
    auto transformed_noisy = (SUM_TYPE*) malloc(transformation_size*sizeof(SUM_TYPE));

    bool* cuda_data;
    cuda_malloc(&cuda_data, columns*parameters->image_count);
    cuda_memcpy_to_gpu(cuda_data, data, columns*parameters->image_count);

    // get noisy data
    bool* data_noisy = (bool*) malloc(columns*parameters->image_count * sizeof(bool));
    std::vector<int> one_indices;
    int less_than_2_ones = 0;
    for (int i = 0; i < parameters->image_count; i++)
    {
        bool* img = (bool*) malloc(columns * sizeof(bool));
        for (int j = 0; j < columns; j++)
        {
            int ind = i*columns + j; // j * columns + i ?
            bool val = data[ind];
            if (val)
                one_indices.push_back(j);
            img[j] = val;
        }
        if (one_indices.size() <= 2)
        {
            less_than_2_ones += 1;
            for(int j = 0; j < columns; j++)
            {
                int ind = i*columns + j;
                data_noisy[ind] = img[j];
            }
            free(img);
            continue;
        }
        //std::random_device rd;
        std::mt19937 generator(i);
        std::uniform_int_distribution<int> u_distribution(0, one_indices.size() - 1);

        int swap_swap_index = u_distribution(generator);
        int swap_index = one_indices[swap_swap_index];

        img[swap_index] = false;

        for(int j = 0; j < columns; j++)
        {
            int ind = i*columns + j;
            data_noisy[ind] = img[j];
        }
        free(img);
        one_indices.clear();
    }
    std::cout << "less_than_2_ones=" << less_than_2_ones << std::endl;
//    for (int i = 0; i < 600; i++)
//    {
//        std::cout << data[i];
//    }
//    std::cout << std::endl;
//    for (int i = 0; i < 600; i++)
//    {
//        std::cout << data_noisy[i];
//    }
//    std::cout << std::endl;

    bool* cuda_data_noisy;
    cuda_malloc(&cuda_data_noisy, columns*parameters->image_count);
    cuda_memcpy_to_gpu(cuda_data_noisy, data_noisy, columns*parameters->image_count);

    uint thread_count = parameters->block_count*parameters->threads_per_block;
    kernel_decorator(
            mult_matrix<SUM_TYPE>,
            parameters->block_count, parameters->threads_per_block, true,
            cuda_transformation, cuda_data, cuda_transformed, rows, columns, parameters->image_count, thread_count
    );

    kernel_decorator(
            mult_matrix<SUM_TYPE>,
            parameters->block_count, parameters->threads_per_block, true,
            cuda_transformation, cuda_data_noisy, cuda_transformed_noisy, rows, columns, parameters->image_count, thread_count
    );

    cuda_memcpy_from_gpu(transformed, cuda_transformed, transformation_size);
    cuda_memcpy_from_gpu(transformed_noisy, cuda_transformed_noisy, transformation_size);

    auto max = transformed[0];
    auto max_i = 0;
    auto min = transformed[0];
    auto min_i = 0;
    for (int i = 1; i < transformation_size; i++)
    {
        auto el = transformed[i];
        if (min > el)
        {
            min_i = i;
            min = el;
        }
        if (max < el)
        {
            max_i = i;
            max = el;
        }
    }
    std::cout <<std::endl;
    std::cout << "max=" << max << " max_i=" << max_i << std::endl;
    std::cout << "min=" << min << " min_i=" << min_i << std::endl << std::endl;

    std::vector<short*> images(parameters->image_count);
    std::vector<short*> images_noisy(parameters->image_count);
    for(int i = 0; i < parameters->image_count; i++)
    {
        auto image = (short*) malloc((rows * sizeof(short)));
        auto image_noisy = (short*) malloc((rows * sizeof(short)));
        for(int j = 0; j < rows; j++)
        {
            auto el = transformed[j*parameters->image_count+i];
            auto el_noisy = transformed_noisy[j*parameters->image_count+i];
            image[j] = el;
            image_noisy[j] = el_noisy;
        }
        images[i] = image;
        images_noisy[i] = image_noisy;
    }

    cuda_free(cuda_transformation);
    cuda_free(cuda_transformed);

    free(transformed);
    free(transformed_noisy);

    SDM_CS1<int, short, short, short> sdm(parameters->mask_length, parameters->address_length,
                                          parameters->value_length, parameters->cells_count, parameters->block_count,
                                          parameters->threads_per_block);

    long write_time_start = clock();
    std::cout << "Started writing ";
    int act_zero = 0;
    std::vector<int> acts(parameters->image_count);
    for(int i = 0; i < parameters->images_read; i++)
    {
        short* image = images[i];
        int act = sdm.write(image);
        act_zero += (act == 0);
        acts[i] = act;
        if ((i+1) % 1000 == 0)
            std::cout << (i+1) << " ";
    }
    //sdm.print_state();
    std::cout << std::endl ;
    long write_time = clock() - write_time_start;

    double sum_l1 = 0;
    long read_time_start = clock();
    std::cout << "Started reading ";
    double sum_l1_arr = 0;

    double max_l1 = 0;
    double max_l1_ind = -1;

    double min_l1 = 1e12;
    double min_l1_ind = -1;

    double avg_l1_r = 0;
    double avg_l1_f = 0;
    double avg_l1_c = 0;
    int read_zeros = 0;
    std::ofstream restored;
    restored.open(output_path + "/cs1_noisy_K_" + std::to_string(parameters->mask_length) +
                  "_I_" + std::to_string(parameters->images_read) + ".csv");

    for(int i = 0; i < parameters->images_read; i++)
    {
        auto el = images[i];
        auto el_noisy = images_noisy[i];
        double l1_arr = 0;
//        if (i == 253)
//        {
//            std::cout << std::endl;
//            for (int j = 0; j < 150; j++)
//            {
//                std::cout << el[j] << ",";
//            }
//            std::cout << std::endl;
//            for (int j = 0; j < 150; j++)
//            {
//                std::cout << el_noisy[j] << ",";
//            }
//            std::cout << std::endl;
////            for (int j = 0; j < 150; j++)
////            {
////                std::cout << remembered[j] << ",";
////            }
////            std::cout << std::endl;
//        }
        double* remembered = sdm.read(el_noisy);
        double l1 = 0;
        double l1_r = 0;
        double l1_f = 0;
        double l1_c = 0;
        std::vector<double> rem_arr(parameters->value_length);
        std::vector<short> img_arr(parameters->value_length);
        bool is_zeros = true;
        for (int j = 0; j < parameters->value_length; j++)
        {
            double rem = remembered[j];
            //std::cout << rem << " ";
            auto sep = (j == parameters->value_length - 1) ? "\n" : ",";
            restored << rem << sep;
            if (abs(rem) > 1e-6)
                is_zeros = false;
            rem_arr[j] = rem;
            auto elj = el[j];
            img_arr[j] = elj;
            double elj_r = round(rem);
            double elj_f = floor(rem);
            double elj_c = ceil(rem);
            l1_r += abs(elj_r - elj);
            l1_f += abs(elj_f - elj);
            l1_c += abs(elj_c - elj);
            l1 += abs(rem - elj);
        }
        if (is_zeros)
        {
            read_zeros += 1;
            continue;
        }
        if (max_l1 < l1)
        {
            max_l1 = l1;
            max_l1_ind = i;
        }
        if (min_l1 > l1)
        {
            min_l1 = l1;
            min_l1_ind = i;
        }

        sum_l1 += l1;
        l1_arr += l1;
        sum_l1_arr += l1_arr;
        avg_l1_r += l1_r / parameters->images_read;
        avg_l1_f += l1_f / parameters->images_read;
        avg_l1_c += l1_c / parameters->images_read;
        free(remembered);
        if ((i+1) % 1000 == 0)
            std::cout << (i+1) << " ";
    }
    std::cout << std::endl;
    long read_time = clock() - read_time_start;
    restored.close();

    report.insert({"act_zero", act_zero});
    report.insert({"avg_l1", sum_l1 / parameters->images_read});
    report.insert({"avg_l1_r", avg_l1_r});
    report.insert({"avg_l1_f", avg_l1_f});
    report.insert({"avg_l1_c", avg_l1_c});
    report.insert({"max_l1", max_l1});
    report.insert({"max_l1_ind", max_l1_ind});
    report.insert({"min_l1", min_l1});
    report.insert({"min_l1_ind", min_l1_ind});
    report.insert({"mae", sum_l1_arr / parameters->images_read / rows});
    report.insert({"avg_read_time", (double)read_time / parameters->images_read});
    report.insert({"avg_write_time", (double)write_time / parameters->images_read});
    report.insert({"read_zeros", read_zeros});
//    report.insert({"min_activations", sdm.get_min_activations()});
//    report.insert({"max_activations", sdm.get_max_activations()});
//    report.insert({"activated_cells_count", sdm.get_activations_num()});

    cuda_free(cuda_data);
    cuda_free(cuda_data_noisy);

    free(data_noisy);

    return report;
}



report_map Runners::CS1Runner::noisy_2(const std::string& data_path, const std::string& output_path)
{
    report_map report;

    report.insert({"mask_length", parameters->mask_length});
    report.insert({"value_length", parameters->value_length});
    report.insert({"address_length", parameters->address_length});
    report.insert({"cells_count", parameters->cells_count});
    report.insert({"image_count", parameters->image_count});
    report.insert({"images_read", parameters->images_read});
    report.insert({"block_count", parameters->block_count});
    report.insert({"threads_per_block", parameters->threads_per_block});
    report.insert({"labels_count", parameters->labels_count});
    report.insert({"target_count", parameters->target_count});
    report.insert({"bits_per_num", parameters->bits_per_num});

    int rows = parameters->target_count;
    int columns = 4*parameters->labels_count;
    bool* transformation = (bool*) malloc(rows*columns*sizeof(bool));
    bool* cuda_transformation;

    cuda_malloc(&cuda_transformation, rows*columns);

    kernel_decorator(
            generate_small_random_matrix<bool>,
            parameters->block_count, parameters->threads_per_block, true,
            rows, columns, cuda_transformation
    );

    cuda_memcpy_from_gpu(transformation, cuda_transformation, rows*columns);

    uint transformation_size = rows*parameters->image_count;

//    std::ofstream transformation_file;
//    auto transformation_file_path = output_path + "/cs1_noisy_2_matrix_K_" + std::to_string(parameters->mask_length) +
//                                    "_I_" + std::to_string(parameters->images_read) + ".csv";
//    transformation_file.open(transformation_file_path);
//
//    for (int i = 0; i < rows; i++)
//    {
//        for (int j = 0; j < columns; j++)
//        {
//            auto ind = i*columns + j;
//            auto el = transformation[ind];
//            int to_write = 2*el - 1;
//            auto sep = (j == columns - 1) ? "\n" : ",";
//            transformation_file << to_write << sep;
//        }
//    }
//    transformation_file.close();

    typedef int SUM_TYPE;

    SUM_TYPE* cuda_transformed;
    cuda_malloc(&cuda_transformed, transformation_size);
    cuda_memset(cuda_transformed, (SUM_TYPE)0, transformation_size);

    SUM_TYPE* cuda_transformed_noisy;
    cuda_malloc(&cuda_transformed_noisy, transformation_size);
    cuda_memset(cuda_transformed_noisy, (SUM_TYPE)0, transformation_size);

    auto transformed = (SUM_TYPE*) malloc(transformation_size*sizeof(SUM_TYPE));
    auto transformed_noisy = (SUM_TYPE*) malloc(transformation_size*sizeof(SUM_TYPE));

    bool* cuda_data;
    cuda_malloc(&cuda_data, columns*parameters->image_count);
    cuda_memcpy_to_gpu(cuda_data, data, columns*parameters->image_count);

    // get noisy data
    bool* data_noisy = (bool*) malloc(columns*parameters->image_count * sizeof(bool));
    std::vector<int> one_indices;
    int less_than_2_ones = 0;
    for (int i = 0; i < parameters->image_count; i++)
    {
        bool* img = (bool*) malloc(columns * sizeof(bool));
        for (int j = 0; j < columns; j++)
        {
            int ind = i*columns + j; // j * columns + i ?
            bool val = data[ind];
            if (val)
                one_indices.push_back(j);
            img[j] = val;
        }
        if (one_indices.size() <= 2)
        {
            less_than_2_ones += 1;
            for(int j = 0; j < columns; j++)
            {
                int ind = i*columns + j;
                data_noisy[ind] = img[j];
            }
            free(img);
            continue;
        }
        //std::random_device rd;
        std::mt19937 generator(i);
        std::uniform_int_distribution<int> u_distribution(0, one_indices.size() - 1);

        int swap_swap_index_1 = u_distribution(generator);
        int swap_index_1 = one_indices[swap_swap_index_1];
        img[swap_index_1] = false;

        int swap_swap_index_2 = u_distribution(generator);
        while (swap_swap_index_2 == swap_swap_index_1)
        {
            swap_swap_index_2 = u_distribution(generator);
        }
        int swap_index_2 = one_indices[swap_swap_index_2];
        img[swap_index_2] = false;


        for(int j = 0; j < columns; j++)
        {
            int ind = i*columns + j;
            data_noisy[ind] = img[j];
        }
        free(img);
        one_indices.clear();
    }
    std::cout << "less_than_2_ones=" << less_than_2_ones << std::endl;
//    for (int i = 0; i < 600; i++)
//    {
//        std::cout << data[i];
//    }
//    std::cout << std::endl;
//    for (int i = 0; i < 600; i++)
//    {
//        std::cout << data_noisy[i];
//    }
//    std::cout << std::endl;

    bool* cuda_data_noisy;
    cuda_malloc(&cuda_data_noisy, columns*parameters->image_count);
    cuda_memcpy_to_gpu(cuda_data_noisy, data_noisy, columns*parameters->image_count);

    uint thread_count = parameters->block_count*parameters->threads_per_block;
    kernel_decorator(
            mult_matrix<SUM_TYPE>,
            parameters->block_count, parameters->threads_per_block, true,
            cuda_transformation, cuda_data, cuda_transformed, rows, columns, parameters->image_count, thread_count
    );

    kernel_decorator(
            mult_matrix<SUM_TYPE>,
            parameters->block_count, parameters->threads_per_block, true,
            cuda_transformation, cuda_data_noisy, cuda_transformed_noisy, rows, columns, parameters->image_count, thread_count
    );

    cuda_memcpy_from_gpu(transformed, cuda_transformed, transformation_size);
    cuda_memcpy_from_gpu(transformed_noisy, cuda_transformed_noisy, transformation_size);

    auto max = transformed[0];
    auto max_i = 0;
    auto min = transformed[0];
    auto min_i = 0;
    for (int i = 1; i < transformation_size; i++)
    {
        auto el = transformed[i];
        if (min > el)
        {
            min_i = i;
            min = el;
        }
        if (max < el)
        {
            max_i = i;
            max = el;
        }
    }
    std::cout <<std::endl;
    std::cout << "max=" << max << " max_i=" << max_i << std::endl;
    std::cout << "min=" << min << " min_i=" << min_i << std::endl << std::endl;

    std::vector<short*> images(parameters->image_count);
    std::vector<short*> images_noisy(parameters->image_count);
    for(int i = 0; i < parameters->image_count; i++)
    {
        auto image = (short*) malloc((rows * sizeof(short)));
        auto image_noisy = (short*) malloc((rows * sizeof(short)));
        for(int j = 0; j < rows; j++)
        {
            auto el = transformed[j*parameters->image_count+i];
            auto el_noisy = transformed_noisy[j*parameters->image_count+i];
            image[j] = el;
            image_noisy[j] = el_noisy;
        }
        images[i] = image;
        images_noisy[i] = image_noisy;
    }

    cuda_free(cuda_transformation);
    cuda_free(cuda_transformed);

    free(transformed);
    free(transformed_noisy);

    SDM_CS1<int, short, short, short> sdm(parameters->mask_length, parameters->address_length,
                                          parameters->value_length, parameters->cells_count, parameters->block_count,
                                          parameters->threads_per_block);

    long write_time_start = clock();
    std::cout << "Started writing ";
    int act_zero = 0;
    std::vector<int> acts(parameters->image_count);
    for(int i = 0; i < parameters->images_read; i++)
    {
        short* image = images[i];
        int act = sdm.write(image);
        act_zero += (act == 0);
        acts[i] = act;
        if ((i+1) % 1000 == 0)
            std::cout << (i+1) << " ";
    }
    //sdm.print_state();
    std::cout << std::endl ;
    long write_time = clock() - write_time_start;

    double sum_l1 = 0;
    long read_time_start = clock();
    std::cout << "Started reading ";
    double sum_l1_arr = 0;

    double max_l1 = 0;
    double max_l1_ind = -1;

    double min_l1 = 1e12;
    double min_l1_ind = -1;

    double avg_l1_r = 0;
    double avg_l1_f = 0;
    double avg_l1_c = 0;
    int read_zeros = 0;
    std::ofstream restored;
    restored.open(output_path + "/cs1_noisy_2_K_" + std::to_string(parameters->mask_length) +
                  "_I_" + std::to_string(parameters->images_read) + ".csv");

    for(int i = 0; i < parameters->images_read; i++)
    {
        auto el = images[i];
        auto el_noisy = images_noisy[i];
        double l1_arr = 0;
//        if (i == 253)
//        {
//            std::cout << std::endl;
//            for (int j = 0; j < 150; j++)
//            {
//                std::cout << el[j] << ",";
//            }
//            std::cout << std::endl;
//            for (int j = 0; j < 150; j++)
//            {
//                std::cout << el_noisy[j] << ",";
//            }
//            std::cout << std::endl;
////            for (int j = 0; j < 150; j++)
////            {
////                std::cout << remembered[j] << ",";
////            }
////            std::cout << std::endl;
//        }
        double* remembered = sdm.read(el_noisy);
        double l1 = 0;
        double l1_r = 0;
        double l1_f = 0;
        double l1_c = 0;
        std::vector<double> rem_arr(parameters->value_length);
        std::vector<short> img_arr(parameters->value_length);
        bool is_zeros = true;
        for (int j = 0; j < parameters->value_length; j++)
        {
            double rem = remembered[j];
            //std::cout << rem << " ";
            auto sep = (j == parameters->value_length - 1) ? "\n" : ",";
            restored << rem << sep;
            if (abs(rem) > 1e-6)
                is_zeros = false;
            rem_arr[j] = rem;
            auto elj = el[j];
            img_arr[j] = elj;
            double elj_r = round(rem);
            double elj_f = floor(rem);
            double elj_c = ceil(rem);
            l1_r += abs(elj_r - elj);
            l1_f += abs(elj_f - elj);
            l1_c += abs(elj_c - elj);
            l1 += abs(rem - elj);
        }
        if (is_zeros)
        {
            read_zeros += 1;
            continue;
        }
        if (max_l1 < l1)
        {
            max_l1 = l1;
            max_l1_ind = i;
        }
        if (min_l1 > l1)
        {
            min_l1 = l1;
            min_l1_ind = i;
        }

        sum_l1 += l1;
        l1_arr += l1;
        sum_l1_arr += l1_arr;
        avg_l1_r += l1_r / parameters->images_read;
        avg_l1_f += l1_f / parameters->images_read;
        avg_l1_c += l1_c / parameters->images_read;
        free(remembered);
        if ((i+1) % 1000 == 0)
            std::cout << (i+1) << " ";
    }
    std::cout << std::endl;
    long read_time = clock() - read_time_start;
    restored.close();

    report.insert({"act_zero", act_zero});
    report.insert({"avg_l1", sum_l1 / parameters->images_read});
    report.insert({"avg_l1_r", avg_l1_r});
    report.insert({"avg_l1_f", avg_l1_f});
    report.insert({"avg_l1_c", avg_l1_c});
    report.insert({"max_l1", max_l1});
    report.insert({"max_l1_ind", max_l1_ind});
    report.insert({"min_l1", min_l1});
    report.insert({"min_l1_ind", min_l1_ind});
    report.insert({"mae", sum_l1_arr / parameters->images_read / rows});
    report.insert({"avg_read_time", (double)read_time / parameters->images_read});
    report.insert({"avg_write_time", (double)write_time / parameters->images_read});
    report.insert({"read_zeros", read_zeros});
    report.insert({"less_than_2_ones", less_than_2_ones});
//    report.insert({"min_activations", sdm.get_min_activations()});
//    report.insert({"max_activations", sdm.get_max_activations()});
//    report.insert({"activated_cells_count", sdm.get_activations_num()});

    cuda_free(cuda_data);
    cuda_free(cuda_data_noisy);

    free(data_noisy);

    return report;
}
#endif //cs1_runner_cu
